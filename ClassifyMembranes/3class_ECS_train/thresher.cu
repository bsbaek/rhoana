
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void
thresh(float *raw, unsigned int *labels, int rawsize,
       unsigned int *packed, int packedsize,
       float thresh, unsigned int offset,
       unsigned int *labelsums)
{
    int bitidx, rawidx;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < packedsize) {
        unsigned int temp, ls;

        temp = 0;
        ls = 0;
        
        for (bitidx = 0; bitidx < sizeof(unsigned int) * 8; bitidx++) {
            rawidx = idx * sizeof(unsigned int) * 8 + bitidx;
            if (rawidx < rawsize) {
                if (raw[rawidx] > thresh) {
                    temp |= (1 << bitidx);
                    ls += labels[rawidx];
                }
            }
        }
        packed[idx + offset] = temp;
        labelsums[idx] = ls;
    }
}

__global__ void
outerprod(unsigned int *packed, int packedsize,
          int N, int sub_x, int sub_y,
          unsigned int *prod)
{
    unsigned int tid = threadIdx.x;
    extern __shared__ unsigned int sdata[];
    int bx = blockIdx.x + sub_x;
    int by = blockIdx.y + sub_y;
    
    if (blockIdx.x < blockIdx.y) {
        if (tid == 0) prod[(bx * N + by)] = 0;
        return;
    }

    unsigned int i = tid + packedsize * bx;
    unsigned int j = tid + packedsize * by;
    unsigned int upper = packedsize * (bx + 1);
    unsigned int mySum = 0;

    while (i < upper)
    {
        mySum += __popc(packed[i] & packed[j]);
        i += 512;
        j += 512;
    }

    // do reduction in shared mem
    sdata[tid] = mySum;
    __syncthreads();

    if (tid < 256) {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }
    __syncthreads();

    if (tid < 128) {
        sdata[tid] = mySum = mySum + sdata[tid + 128];
    }
    __syncthreads();

    if (tid <  64) {
        sdata[tid] = mySum = mySum + sdata[tid +  64];
    }
    __syncthreads();

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile unsigned int *smem = sdata;

        smem[tid] = mySum = mySum + smem[tid + 32];
        smem[tid] = mySum = mySum + smem[tid + 16];
        smem[tid] = mySum = mySum + smem[tid +  8];
        smem[tid] = mySum = mySum + smem[tid +  4];
        smem[tid] = mySum = mySum + smem[tid +  2];
        smem[tid] = mySum = mySum + smem[tid +  1];
    }

    // write result for this block to outer product
    if (tid == 0) {
        prod[bx * N + by] = mySum;
    }
}
